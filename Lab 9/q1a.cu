
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add1(int *a, int *b, int *c) {
c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
int main1(void) {
int n;
printf("Enter n:");
scanf("%d",&n);
int a[n], b[n], c[n];
int *d_a, *d_b, *d_c;
int size =n*sizeof(int);
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
for(int i=0;i<n;i++)
	scanf("%d",&a[i]);
for(int i=0;i<n;i++)
	scanf("%d",&b[i]);
hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
add1<<<n,1>>>(d_a, d_b, d_c);
// Copy result back to host
hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
//print result
for(int i=0;i<n;i++)
	printf("%d ",c[i]);
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
