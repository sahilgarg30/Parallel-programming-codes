
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void mul2(int *a, int *b, int *c,int col) {
int x;
for(int i=0;i<blockDim.x;i++){
	x=0;
	for(int j=0;j<col;j++){
		x+= a[i*col+j] *b[j*col+threadIdx.x];
	}
	c[i*col+ threadIdx.x]=x;
}
}
int main10(void) {
int r,col;
printf("Enter dimensions:");
scanf("%d %d",&r,&col);
int a[r*col], b[r*col], c[r*col];
int *d_a, *d_b, *d_c;
int size = r*col*sizeof(int);
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
printf("Enter the 2 matrices: \n");
for(int i=0;i<r*col;i++)
	scanf("%d",&a[i]);
for(int i=0;i<r*col;i++)
	scanf("%d",&b[i]);
hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
mul2<<<1,r>>>(d_a, d_b, d_c,col);
// Copy result back to host
hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
//print result
for(int i=0;i<r*col;i++){
	printf("%d ",c[i]);
}
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
