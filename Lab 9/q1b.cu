
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add2(int *a, int *b, int *c) {
c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}
int main2(void) {
int n;
printf("Enter n:");
scanf("%d",&n);
int a[n], b[n], c[n];
int *d_a, *d_b, *d_c;
int size =n*sizeof(int);
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
for(int i=0;i<n;i++)
	scanf("%d",&a[i]);
for(int i=0;i<n;i++)
	scanf("%d",&b[i]);
hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
add2<<<1,n>>>(d_a, d_b, d_c);
// Copy result back to host
hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
//print result
for(int i=0;i<n;i++)
	printf("%d ",c[i]);
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
