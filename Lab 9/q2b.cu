
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add3(int *a, int *b, int *c,int r) {
for(int i=0;i<r;i++)
c[i*blockDim.x+ threadIdx.x] = a[i*blockDim.x+ threadIdx.x] + b[threadIdx.x+i*blockDim.x];
}
int main4(void) {
int r,col;
printf("Enter dimensions:");
scanf("%d %d",&r,&col);
int a[r*col], b[r*col], c[r*col];
int *d_a, *d_b, *d_c;
int size = r*col*sizeof(int);
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
for(int i=0;i<r*col;i++)
	scanf("%d",&a[i]);
for(int i=0;i<r*col;i++)
	scanf("%d",&b[i]);
hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
add3<<<1,col>>>(d_a, d_b, d_c,r);
// Copy result back to host
hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
//print result
for(int i=0;i<r*col;i++){
	printf("%d ",c[i]);
}
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
