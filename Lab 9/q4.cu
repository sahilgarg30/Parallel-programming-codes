
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void trans(int *a, int *c,int n) {
c[blockIdx.x*n + threadIdx.x] = a[threadIdx.x*n + blockIdx.x];
}

int main(void) {
int n;
printf("Enter n:");
scanf("%d",&n);
int a[n*n], b[n*n];
int *d_a, *d_b;
int size = n*n*sizeof(int);
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
for(int i=0;i<n*n;i++)
	scanf("%d",&a[i]);
hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
trans<<<n,n>>>(d_a, d_b,n);
hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
for(int i=0;i<n;i++){
	for(int j=0;j<n;j++)
	printf("%d ",b[i*n+j]);
	printf("\n");
}
hipFree(d_a);
hipFree(d_b);
return 0;
}
