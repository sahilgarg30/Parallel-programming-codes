
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add4(int *a, int *b, int *c) {
c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}
int main5(void) {
int r,col;
printf("Enter dimensions:");
scanf("%d %d",&r,&col);
int a[r*col], b[r*col], c[r*col];
int *d_a, *d_b, *d_c;
int size = r*col*sizeof(int);
// Allocate space for device copies of a, b, c
hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);
for(int i=0;i<r*col;i++)
	scanf("%d",&a[i]);
for(int i=0;i<r*col;i++)
	scanf("%d",&b[i]);
hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
add4<<<1,r*col>>>(d_a, d_b, d_c);
// Copy result back to host
hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
//print result
for(int i=0;i<r*col;i++){
	printf("%d ",c[i]);
}
// Cleanup
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
