
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
__global__ void add(char *a,int size1) {
	int id = blockIdx.x;
	char temp = a[id];
	a[id] = a[size1-id-1];
	a[size1-id-1] = temp;
}

int main2(void) {
char a[50];
char *d_a;
scanf("%s",a);

int size1 = sizeof(char)*strlen(a);

hipMalloc((void **)&d_a, size1);

hipMemcpy(d_a,a, size1, hipMemcpyHostToDevice);
add<<<size1/2,1>>>(d_a,size1);
hipMemcpy(a,d_a,size1, hipMemcpyDeviceToHost);
printf("%s",a);
hipFree(d_a);

return 0;
}
