
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
__global__ void add(char *a, char *b,int size1,int size2) {
	int id = blockIdx.x;
	if(id+size2 > size1) return;
	int i;
	for(i=0;i<size2;i++){
		if(a[id+i]!=b[i]) break;
	}
	if(i==size2) printf("%d ",id);
}
int main1(void) {
char a[50],b[50];
char *d_a, *d_b;
scanf("%s %s",a,b);

int size1 = sizeof(char)*strlen(a);
int size2 = sizeof(char)*strlen(b);

hipMalloc((void **)&d_a, size1);
hipMalloc((void **)&d_b, size2);

hipMemcpy(d_a, a, size1, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size2, hipMemcpyHostToDevice);
add<<<size1,1>>>(d_a, d_b,size1,size2);
hipFree(d_a);
hipFree(d_b);
return 0;
}
