
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
__global__ void bin(char *a,int size1) {
	int id = blockIdx.x;
	int val = a[id],res = 0,x = 1;
	while(val>0){
		res += x*(val%2);
		x *=10;
		val /=2;
	}
	printf("%d - %d\n",id,res);
}

int main3(void) {
char a[50];
char *d_a;
scanf("%s",a);

int size1 = sizeof(char)*strlen(a);

hipMalloc((void **)&d_a, size1);

hipMemcpy(d_a,a, size1, hipMemcpyHostToDevice);
bin<<<size1,1>>>(d_a,size1);
hipFree(d_a);
return 0;
}
