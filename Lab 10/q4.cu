#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
__global__ void add(char *a, char *b,int size1,int size2,int *c) {
	int id = blockIdx.x;
	if(id+size2 > size1) return;
	int i;
	for(i=0;i<size2;i++){
		if(a[id+i]!=b[i]) break;
	}
	if(i==size2 && (id==0 || a[id-1]==' ') && (id+size2 == size1 || a[id+size2]==' ')){
		c[id] = 1;
		return;
	}
	c[id] = 0;
}
int main4(void) {
char a[50],b[50];
char *d_a, *d_b;
int c[50]; int *d_c;
gets(a);
scanf("%s",b);

int size1 = sizeof(char)*strlen(a);
int size2 = sizeof(char)*strlen(b);

hipMalloc((void **)&d_a, size1);
hipMalloc((void **)&d_b, size2);
hipMalloc((void **)&d_c,sizeof(int)*size1);

hipMemcpy(d_a, a, size1, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size2, hipMemcpyHostToDevice);
add<<<size1,1>>>(d_a, d_b,size1,size2,d_c);
hipMemcpy(c,d_c, sizeof(int)*size1, hipMemcpyDeviceToHost);
int cs = 0;
for(int i=0;i<size1;i++) if(c[i]==1) cs++;
printf("%d",cs);
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
return 0;
}
