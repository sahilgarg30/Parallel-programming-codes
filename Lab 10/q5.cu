
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
__global__ void add(int *a,int *b,int *c,int n) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	c[x*n+y] = a[x*n+y]+b[x*n+y];
}

__global__ void multiply(int *a,int *b,int *c,int n) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int val = 0;
	for(int i=0;i<n;i++){
		val += a[x*n+i]*b[i*n+y];
	}
	c[x*n+y] = val;
}

int main(void) {
int a[50],b[50],c[50],d[50];
int *d_a,*d_b,*d_c,*d_d;

int n;
scanf("%d",&n);

for(int i=0;i<n;i++) for(int j=0;j<n;j++) scanf("%d",&a[i*n+j]);
for(int i=0;i<n;i++) for(int j=0;j<n;j++) scanf("%d",&b[i*n+j]);


hipMalloc((void **)&d_a,sizeof(int)*n*n);
hipMalloc((void **)&d_b,sizeof(int)*n*n);
hipMalloc((void **)&d_c,sizeof(int)*n*n);
hipMalloc((void **)&d_d,sizeof(int)*n*n);

hipMemcpy(d_a, a, n*n*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_b, b, n*n*sizeof(int), hipMemcpyHostToDevice);
dim3 d1(n,n,1);
dim3 d2(1,1,1);
add<<<d1,d2>>>(d_a,d_b,d_c,n);
hipMemcpy(c,d_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);
for(int i=0;i<n;i++){
	for(int j=0;j<n;j++) printf("%d ",c[i*n+j]);
	printf("\n");
}

multiply<<<d1,d2>>>(d_a,d_b,d_d,n);
hipMemcpy(d,d_d, sizeof(int)*n*n, hipMemcpyDeviceToHost);
for(int i=0;i<n;i++){
	for(int j=0;j<n;j++) printf("%d ",d[i*n+j]);
	printf("\n");
}
hipFree(d_a);
hipFree(d_b);
hipFree(d_c);
hipFree(d_d);
return 0;
}
